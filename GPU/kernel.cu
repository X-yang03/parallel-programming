#include "hip/hip_runtime.h"
﻿
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include <iomanip>
#include"hip/hip_runtime.h"
#include""
#include"hip/device_functions.h"
#include<windows.h>

using namespace std;
const int N = 1024;
const int BLOCK_SIZE = 1024;
float **A;
float elm[N][N] = { 0 };

float* result = new float[N * N];
float* temp = new float[N * N];

long long head, tail, freq;

void A_init() {     //矩阵的初始化
    A = new float* [N];
    for (int i = 0; i < N; i++) {
        A[i] = new float[N];
    }
    for (int i = 0; i < N; i++) {
        A[i][i] = 1.0;
        for (int j = i + 1; j < N; j++) {
            A[i][j] = rand() % 5000;
        }

    }
    for (int k = 0; k < N; k++) {
        for (int i = k + 1; i < N; i++) {
            for (int j = 0; j < N; j++) {
                A[i][j] += A[k][j];
                A[i][j] = (int)A[i][j] % 5000;
            }
        }
    }
}

void copy_matrix() {
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			temp[i * N + j] = A[i][j];
		}
	}
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            elm[i][j] = A[i][j];
        }
    }
}

void transfer_result() {
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			elm[i][j] = result[i * N + j];
		}
	}
}

void print_result() {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout << elm[i][j] << " ";
        }
        cout << endl;
    }
}

void LU() {    //普通消元算法
    for (int k = 0; k < N; k++) {
        for (int j = k + 1; j < N; j++) {
            elm[k][j] = elm[k][j] / elm[k][k];
        }
        elm[k][k] = 1.0;

        for (int i = k + 1; i < N; i++) {
            for (int j = k + 1; j < N; j++) {
                elm[i][j] = elm[i][j] - elm[i][k] * elm[k][j];
            }
            elm[i][k] = 0;
        }
    }
}


__global__ void division_kernel(float* data, int k, int N) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;//计算线程索引
    int element = data[k * N + k];
    int temp = data[k * N + tid];
    data[k * N + tid] = (float)temp / element;
    return;
}

__global__ void eliminate_kernel(float* data, int k, int N) {
    int tx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tx == 0)
        data[k * N + k] = 1.0;//对角线元素设为 1
    int row = k + 1 + blockIdx.x;//每个块负责一行
    while (row < N) {
        int tid = threadIdx.x;
        while (k + 1 + tid < N) {
            int col = k + 1 + tid;
            float temp_1 = data[(row * N) + col];
            float temp_2 = data[(row * N) + k];
            float temp_3 = data[k * N + col];
            data[(row * N) + col] = temp_1 - temp_2 * temp_3;
            tid = tid + blockDim.x;
        }
        __syncthreads();//块内同步
        if (threadIdx.x == 0) {
            data[row * N + k] = 0;
        }
        row += gridDim.x;
    }
    return;
}

int main() {
    A_init();
    copy_matrix();
    QueryPerformanceFrequency((LARGE_INTEGER*)&freq);

    QueryPerformanceCounter((LARGE_INTEGER*)&head);
    LU();
    QueryPerformanceCounter((LARGE_INTEGER*)&tail);
    cout << "CPU_LU:" << (tail - head) * 1000 / freq << "ms" << endl;
    print_result();       //平凡消去的执行和时间测试

	hipError_t ret;//用于错误检查，当 CUDA 接口调用成功会返回 cudaSucess
	float* gpudata;
	int size = N * N * sizeof(float);
	ret = hipMalloc(&gpudata, size);//分配显存空间
	if (ret != hipSuccess) {
		printf("hipMalloc gpudata failed!\n");

	}
	ret = hipMemcpy(gpudata, temp, size, hipMemcpyHostToDevice);//将数据传输至 GPU 端
	if (ret != hipSuccess) {
		printf("hipMemcpyHostToDevice failed!\n");
	}

	dim3 dimBlock(BLOCK_SIZE, 1);//线程块
	dim3 dimGrid(1, 1);//线程网格

	hipEvent_t start, stop;//计时器
	float elapsedTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);//开始计时

	for (int k = 0; k < N; k++) {
		division_kernel << <dimGrid, dimBlock >> > (gpudata, k, N);//负责除法任务的核函数
		hipDeviceSynchronize();//CPU 与 GPU 之间的同步函数
		ret = hipGetLastError();
		if (ret != hipSuccess) {
			printf("division_kernel failed, %s\n", hipGetErrorString(ret));
		}
		eliminate_kernel << <dimGrid, dimBlock >> > (gpudata, k,N);//负责消去任务的核函数
		hipDeviceSynchronize();
		ret = hipGetLastError();
		if (ret != hipSuccess) {
			printf("eliminate_kernel failed, %s\n", hipGetErrorString(ret));
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);//停止计时
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU_LU:%f ms\n", elapsedTime);
	hipError_t cudaStatus2 = hipGetLastError();
	if (cudaStatus2 != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus2));
	}
	ret = hipMemcpy(result, gpudata, size, hipMemcpyDeviceToHost);//将数据传回 CPU 端
	if (ret != hipSuccess) {
		printf("hipMemcpyDeviceToHost failed!\n");
	}
	transfer_result();
    print_result();
	hipFree(gpudata);//释放显存空间，用 CUDA 接口分配的空间必须用 hipFree 释放
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
